
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

using std::cout;
using std::cin;
using std::endl;

__global__ void add(const int *a, const int *b, int *c, int len) {
    uint32_t i = threadIdx.x;
    if (i < len) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    // 写一个向量加法
    /*
     * C = A + B
     * 其中A、B、C均为长度为10的向量
     *
     * */
    int length = 10;
    int a[length], b[length], c[length];
    int *a_gpu, *b_gpu, *c_gpu;

    for (int i = 0; i < length; ++i) {
        a[i] = i;
        b[i] = i * i;
    }
    hipMalloc((void **) &a_gpu, length * sizeof(int));
    hipMalloc((void **) &b_gpu, length * sizeof(int));
    hipMalloc((void **) &c_gpu, length * sizeof(int));

    hipMemcpy(a_gpu, a, length * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, length * sizeof(int), hipMemcpyHostToDevice);

    add<<<1, length>>>(a_gpu, b_gpu, c_gpu, length);

    hipMemcpy(c, c_gpu, length * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < length; ++i) {
        cout << i << " ";
    }
    cout << endl;

    return 0;
}