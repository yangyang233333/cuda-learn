
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

using std::cout;
using std::cin;
using std::endl;

__global__ void reduce_sum2(const float *a, float *result) {
    int tid = threadIdx.x;

    result[0] = 0;
    __syncthreads();
    atomicAdd(result, a[tid]);
}

int main() {
    // 写一个数组规约求和，使用原子操作版本
    /*
     * A 是一个长度16的数组，对其进行求和
     * */

    float a[16];
    for (int i = 0; i < 16; ++i) {
        a[i] = (float) i;
    }

    float *a_gpu;
    hipMalloc((void **) &a_gpu, 16 * sizeof(float));
    hipMemcpy(a_gpu, a, 16 * sizeof(float), hipMemcpyHostToDevice);

    float *result_gpu;

    hipMalloc((void **) &result_gpu, 1 * sizeof(float));
    reduce_sum2<<<1, 16>>>(a_gpu, result_gpu);
    float result = 0;
    hipMemcpy(&result, result_gpu, 1 * sizeof(float), hipMemcpyDeviceToHost);
    cout << "result=" << result << endl;
}