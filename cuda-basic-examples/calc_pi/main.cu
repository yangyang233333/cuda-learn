
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

using std::cout;
using std::cin;
using std::endl;

//x,y 表示点的坐标，num表示点的数量，计算出result
//result长度为num，如果(x_i, y_i)在圆内，则result[i]为1，反之为0
__global__ void calc_num(const double *x, const double *y, int *result, int num) {
    for (auto thread_id = blockIdx.x * blockDim.x + threadIdx.x;
         thread_id < num; thread_id += blockDim.x * gridDim.x) {
        // 计算点到圆心的距离
        double distance = (x[thread_id] - 1) * (x[thread_id] - 1) + (y[thread_id] - 1) * (y[thread_id] - 1);
        if (distance < 1) {
            result[thread_id] = 1;
        } else {
            result[thread_id] = 0;
        }
    }
}

// 归约求和
__global__ void reduce_sum(const int *array, int *array_sum, int N) {
    int thread_id = threadIdx.x;
    array_sum[0] = 0;
    __shared__ int shared_data[512];

    for (int count = 0; count < ceilf(N / 512); ++count) {
        if (thread_id + count * 512 < N) {
            shared_data[thread_id] = array[thread_id];
            __syncthreads();
        }
        for (int i = 256; i > 0; i /= 2) {
            if (thread_id < i && thread_id + count * 512 < N) {
                shared_data[thread_id] = shared_data[thread_id] + shared_data[thread_id + i];
            }
            __syncthreads();
        }
        if (thread_id == 0) {
            array_sum[0] += shared_data[0];
        }
    }
}

int main() {
    // 写一个计算圆周率PI的kernel
    /*
     * 生成N个点，分别放入x[]和y[]，然后计算(x, y)到圆心的距离，
     * */
    constexpr int N = 100000000;
    srand(time(nullptr));
    auto x = new double[N];
    auto y = new double[N];
    for (int i = 0; i < N; ++i) {
        x[i] = rand() % 10000 / 10000.;
        y[i] = rand() % 10000 / 10000.;
    }
    double *x_gpu, *y_gpu;
    hipMalloc((void **) &x_gpu, N * sizeof(float));
    hipMalloc((void **) &y_gpu, N * sizeof(float));
    hipMemcpy(x_gpu, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_gpu, y, N * sizeof(float), hipMemcpyHostToDevice);

    int thread_num = 1024;
    int block_num = 512;
    int *result_gpu;
    hipMalloc((void **) &result_gpu, N * sizeof(int));
    calc_num<<<block_num, thread_num>>>(x_gpu, y_gpu, result_gpu, N);

    int *cnt_gpu;
    hipMalloc((void **) &cnt_gpu, 1 * sizeof(int));
    reduce_sum<<<1, 512>>>(result_gpu, cnt_gpu, N);
    int *cnt = new int;
    hipMemcpy(cnt, cnt_gpu, sizeof(int), hipMemcpyDeviceToHost);

    cout << "pi=" << 4.0 * (1.0 * (*cnt) / N) << endl;

}