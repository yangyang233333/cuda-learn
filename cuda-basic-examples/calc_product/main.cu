
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <numeric>

using std::cout;
using std::cin;
using std::endl;

#define LENGTH 16
#define THREAD_NUM 4
#define BLOCK_NUM 2

__global__ void dot_product(float *a_gpu, float *b_gpu, float *result_gpu) {
    __shared__ float shared_data[THREAD_NUM];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int global_id = tid + bid * blockDim.x;
    while (global_id < LENGTH) {
        shared_data[tid] += a_gpu[global_id] + b_gpu[global_id];
        global_id += THREAD_NUM * BLOCK_NUM;
    }
    __syncthreads();
    // 归约求和
    for (int i = LENGTH / 2; i > 0; i /= 2) {
        if (tid < i) {
            shared_data[tid] = shared_data[tid] + shared_data[tid + i];
        }
        __syncthreads();
    }
    if (tid == 0) {
        result_gpu[bid] = shared_data[0];
    }
}

int main() {
    // 多个block的归约求点积
    /*
     *
     * */

    float a[LENGTH];
    float b[LENGTH];
    for (int i = 0; i < LENGTH; ++i) {
        a[i] = i * (i + 1);
        b[i] = i * (i - 2);
    }

    float *a_gpu, *b_gpu;
    hipMalloc((void **) &a_gpu, LENGTH * sizeof(float));
    hipMalloc((void **) &b_gpu, LENGTH * sizeof(float));
    hipMemcpy(a_gpu, a, LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, LENGTH * sizeof(float), hipMemcpyHostToDevice);

    float *result_gpu;
    hipMalloc((void **) &result_gpu, BLOCK_NUM * sizeof(float));
    // 假设线程数小于LENGTH，模拟需要多个block的场景
    dot_product<<<BLOCK_NUM, THREAD_NUM>>>(a_gpu, b_gpu, result_gpu);

    float result[BLOCK_NUM];
    hipMemcpy(result, result_gpu, BLOCK_NUM * sizeof(float), hipMemcpyDeviceToHost);

    cout << "result=" << std::accumulate(std::begin(result), std::end(result), 0.0) << endl;

}