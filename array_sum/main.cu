
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

using std::cout;
using std::cin;
using std::endl;

__global__ void reduce_sum(float *a, float *result) {
    __shared__ float shared_data[16];
    shared_data[threadIdx.x] = a[threadIdx.x];
    __syncthreads(); // 保证所有线程都完成复制

    for (int i = 8; i > 0; i /= 2) {
        shared_data[threadIdx.x] = shared_data[threadIdx.x] + shared_data[threadIdx.x + i];
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        result[threadIdx.x] = shared_data[threadIdx.x];
    }
}

int main() {
    // 写一个数组规约求和
    /*
     * A 是一个长度16的数组，对其进行求和
     * */

    float a[16];
    for (int i = 0; i < 16; ++i) {
        a[i] = (float) i;
    }

    float *a_gpu;
    hipMalloc((void **) &a_gpu, 16 * sizeof(float));
    hipMemcpy(a_gpu, a, 16 * sizeof(float), hipMemcpyHostToDevice);

    float *result_gpu;

    hipMalloc((void **) &result_gpu, 1 * sizeof(float));
    reduce_sum<<<1, 16>>>(a_gpu, result_gpu);
    float result = 0;
    hipMemcpy(&result, result_gpu, 1 * sizeof(float), hipMemcpyDeviceToHost);
    cout << "result=" << result << endl;
}