
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

using std::cout;
using std::cin;
using std::endl;

__global__ void reduce_sum(const float *a, float *result) {
    // 动态分配的显存，具体大小在调用时指定，例如reduce_sum<<<1, 16, 16>>>
    // 最后一个16就是指的动态显存sh的长度
    extern __shared__ float sh[];
    float *shared_data = sh;
    shared_data[threadIdx.x] = a[threadIdx.x];
    __syncthreads(); // 保证所有线程都完成复制

    for (int i = 8; i > 0; i /= 2) {
        shared_data[threadIdx.x] = shared_data[threadIdx.x] + shared_data[threadIdx.x + i];
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        result[threadIdx.x] = shared_data[threadIdx.x];
    }
}

int main() {
    // 写一个数组规约求和，使用动态分配的显存
    /*
     * A 是一个长度16的数组，对其进行求和
     * */

    float a[16];
    for (int i = 0; i < 16; ++i) {
        a[i] = (float) i;
    }

    float *a_gpu;
    hipMalloc((void **) &a_gpu, 16 * sizeof(float));
    hipMemcpy(a_gpu, a, 16 * sizeof(float), hipMemcpyHostToDevice);

    float *result_gpu;

    hipMalloc((void **) &result_gpu, 1 * sizeof(float));
    reduce_sum<<<1, 16, 16>>>(a_gpu, result_gpu);
    float result = 0;
    hipMemcpy(&result, result_gpu, 1 * sizeof(float), hipMemcpyDeviceToHost);
    cout << "result=" << result << endl;
}